#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cudaSkinning.h"

#include <hip/hip_runtime.h>
#define GLM_FORCE_PURE
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/transform.hpp>
#include <glm/gtx/compatibility.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <stdio.h>

#define PI 3.14159265358979323846

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time);
__global__ void triangleTransformKernel(int numTriangles, glm::mat4 *transformsIn, glm::mat4 *transformsOut, const float time);
__global__ void triangleSimKernel(int numVerts, glm::mat4 *transforms, cudaVertex *in, cudaVertex *out, const float time);
__global__ void skinKernel(int numVerts, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *in, cudaVertex *out, const float time);
__global__ void morphTargetKernel(int numVerts, cudaVertex *target1, cudaVertex *target2, cudaVertex *out, const float alpha);
void cudaFakeSkin(int numVerts, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **) &dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **) &dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	fakeSkinKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);
	hipFree(dv_idata);
	hipFree(dv_odata);
}

void cudaSkin(int numVerts, int numTransforms, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **)&dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **)&dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	glm::mat4 *dv_transforms;
	hipMalloc((void **)&dv_transforms, numTransforms * sizeof(glm::mat4));
	hipMemcpy(dv_transforms, transforms, numTransforms * sizeof(glm::mat4), hipMemcpyHostToDevice);

	cudaVertexBoneData *dv_bones;
	hipMalloc((void **)&dv_bones, numVerts * sizeof(cudaVertexBoneData));
	hipMemcpy(dv_bones, bones, numVerts * sizeof(cudaVertexBoneData), hipMemcpyHostToDevice);

	skinKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_transforms, dv_bones, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);

	hipFree(dv_idata);
	hipFree(dv_odata);
	hipFree(dv_transforms);
	hipFree(dv_bones);
}

void cudaMorph(int numVerts, cudaVertex *target1, cudaVertex *target2, cudaVertex *vertsOut, const float alpha) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_itarget1, *dv_itarget2, *dv_odata;

	hipMalloc((void **) &dv_itarget1, numVerts * sizeof(cudaVertex));
	hipMalloc((void **) &dv_itarget2, numVerts * sizeof(cudaVertex));
	hipMalloc((void **) &dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_itarget1, target1, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);
	hipMemcpy(dv_itarget2, target2, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	morphTargetKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_itarget1, dv_itarget2, dv_odata, alpha);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);
	hipFree(dv_itarget1);
	hipFree(dv_itarget2);
	hipFree(dv_odata);
}

void triangleSim(int numVerts, glm::mat4 *transformsIn, glm::mat4 *transformsOut, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {

	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **)&dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **)&dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);


	glm::mat4 *dv_itransforms, *dv_otransforms;
	hipMalloc((void **)&dv_itransforms, (numVerts / 3) * sizeof(glm::mat4));
	hipMalloc((void **)&dv_otransforms, (numVerts / 3) * sizeof(glm::mat4));
	hipMemcpy(dv_itransforms, transformsIn, (numVerts / 3) * sizeof(glm::mat4), hipMemcpyHostToDevice);
	triangleTransformKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts / 3, dv_itransforms, dv_otransforms, time);
	triangleSimKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_itransforms, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);
	hipMemcpy(transformsOut, dv_otransforms, numVerts / 3 * sizeof(glm::mat4), hipMemcpyDeviceToHost);
	hipFree(dv_idata);
	hipFree(dv_odata);
	hipFree(dv_itransforms);
	hipFree(dv_otransforms);
}

__forceinline__
__device__ __host__
double convertToRadians(double deg) {
	return deg * (PI / 180.);
}

__forceinline__
__device__ __host__
glm::vec3 xm2vec3(XMFLOAT3 other) {
	return glm::vec3(other.x, other.y, other.z);
}

__forceinline__
__device__ __host__
glm::mat4 xm2mat4(XMFLOAT4X4 other) {
	float aaa[16];
	aaa[0] = other._11;
	aaa[1] = other._21;
	aaa[2] = other._31;
	aaa[3] = other._41;

	aaa[4] = other._12;
	aaa[5] = other._22;
	aaa[6] = other._32;
	aaa[7] = other._42;

	aaa[8] = other._13;
	aaa[9] = other._23;
	aaa[10] = other._33;
	aaa[11] = other._43;

	aaa[12] = other._14;
	aaa[13] = other._24;
	aaa[14] = other._34;
	aaa[15] = other._44;
	return glm::make_mat4(aaa);
}

__forceinline__
__device__ __host__
cudaVertex makeCUDAVertex(glm::vec3 position, glm::vec3 normal) {
	cudaVertex ret;
	ret.position.x = position.x;
	ret.position.y = position.y;
	ret.position.z = position.z;
	ret.normal.x = normal.x;
	ret.normal.y = normal.y;
	ret.normal.z = normal.z;
	return ret;
}

__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v = in[i];
	//glm::vec3 pos = xm2vec3(v.position);
	//glm::vec3 nor = xm2vec3(v.normal);
	//auto rot = glm::rotate((time / 24.f) * 360.f, glm::vec3(0, 1, 0));
	//rot = glm::mat4(0.0f);
	//pos = glm::vec3(rot * glm::vec4(pos, 1.f));
	out[i] = in[i];
}

__global__ void skinKernel(int numVerts, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v = in[i];
	auto &bone = bones[i];
	glm::vec3 pos = xm2vec3(v.position);
	glm::vec3 nor = xm2vec3(v.normal);
	glm::mat4 transform(0.f);
	float totalWeight = 0.f;
	for (int k = 0; k < 4; k++) {
		int ID = bone.IDs[k];
		float weight = bone.Weights[k];
		totalWeight += weight;
		glm::mat4 mat = (transforms[ID]);
		transform += mat * weight;
	}

	pos = glm::vec3(transform * glm::vec4(pos, 1.0f));
	nor = glm::vec3(transform * glm::vec4(nor, 0.0f));

	out[i] = makeCUDAVertex(pos, nor);
}

	__global__ void morphTargetKernel(int numVerts, cudaVertex *target1, cudaVertex *target2, cudaVertex *out, float alpha) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v1 = target1[i];
	auto &v2 = target2[i];
	glm::vec3 pos1 = xm2vec3(v1.position);
	glm::vec3 pos2 = xm2vec3(v2.position);
	glm::vec3 nor1 = xm2vec3(v1.normal);
	glm::vec3 nor2 = xm2vec3(v2.normal);
	glm::vec3 pos = glm::lerp(pos1, pos2, alpha);
	glm::vec3 nor = glm::lerp(nor1, nor2, alpha);
	out[i] = makeCUDAVertex(pos, nor);
}
__global__ void triangleTransformKernel(int numTriangles, glm::mat4 *transformsIn, glm::mat4 *transformsOut, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numTriangles) { return; }
	//auto rot = glm::rotate(5, glm::vec3(0, 1, 0));
	glm::vec3 t = glm::vec3(0.1, 0, 0);
	transformsOut[i] = transformsIn[i];
	//transformsOut[i] = glm::translate(transformsIn[i], t);
	//transformsOut[i] = glm::rotate(transformsIn[i], time, glm::vec3(glm::normalize(transformsIn[i][3])));
}

//Twist function from  http://www.iquilezles.org/www/articles/distfunctions/distfunctions.htm


__forceinline__ __device__  glm::vec3 invertSpace(glm::vec3 p, float s)
{
	return s * p / glm::dot(p, p);
}
__forceinline__ __device__ glm::vec3 twist(glm::vec3 p, float time) {
	float t = glm::sin(time) * p.y;
	float ct = glm::cos(t) * 1.0;
	float st = glm::sin(t) * 1.0;

	glm::vec3 pos = p;

	pos.x = p.x * ct - p.z * st;
	pos.z = p.x * st + p.z * ct;
	return pos;
}

__forceinline__ __device__ float fract(float a) {
	float x = floor(a);
	return a - x;
}


__forceinline__ __device__ float lerp(float a, float b, float t) {
	return a * (1.0f - t) + b * t;
}

__forceinline__ __device__  glm::vec4 lerp(glm::vec4 a, glm::vec4 b, float t) {
	return a * (1.0f - t) + b * t;
}

__forceinline__ __device__  float cerp(float a, float b, float t) {
	float cos_t = (1.0f - cos(t*3.14159f)) * 0.5f;
	return lerp(a, b, cos_t);
}

__forceinline__ __device__ glm::vec3 palette(float t, glm::vec3 a, glm::vec3 b, glm::vec3 c, glm::vec3 d)
{
	return a + b * cos(6.28318f *(c*t + d));
}

__forceinline__ __device__ float random(float a, float b, float c) {
	return fract(glm::sin(glm::dot(glm::vec3(a, b, c), glm::vec3(12.9898, 78.233, 578.233)))*43758.5453);
}


__forceinline__ __device__ float interpolateNoise(float x, float y, float z) {
	float x0, y0, z0, x1, y1, z1;

	// Find the grid voxel that this point falls in
	x0 = floor(x);
	y0 = floor(y);
	z0 = floor(z);

	x1 = x0 + 1.0;
	y1 = y0 + 1.0;
	z1 = z0 + 1.0;

	// Generate noise at each of the 8 points
	float FUL, FUR, FLL, FLR, BUL, BUR, BLL, BLR;

	// front upper left
	FUL = random(x0, y1, z1);

	// front upper right
	FUR = random(x1, y1, z1);

	// front lower left
	FLL = random(x0, y0, z1);

	// front lower right
	FLR = random(x1, y0, z1);

	// back upper left
	BUL = random(x0, y1, z0);

	// back upper right
	BUR = random(x1, y1, z0);

	// back lower left
	BLL = random(x0, y0, z0);

	// back lower right
	BLR = random(x1, y0, z0);

	// Find the interpolate t values
	float n0, n1, m0, m1, v;
	float tx = fract(x - x0);
	float ty = fract(y - y0);
	float tz = fract(z - z0);
	tx = (x - x0);
	ty = (y - y0);
	tz = (z - z0);

	// interpolate along x and y for back
	n0 = cerp(BLL, BLR, tx);
	n1 = cerp(BUL, BUR, tx);
	m0 = cerp(n0, n1, ty);

	// interpolate along x and y for front
	n0 = cerp(FLL, FLR, tx);
	n1 = cerp(FUL, FUR, tx);
	m1 = cerp(n0, n1, ty);

	// interpolate along z
	v = cerp(m0, m1, tz);

	return v;
}

__forceinline__ __device__ float generateNoise(float x, float y, float z) {
	float total = 0.0;
	float persistence = 1.0 / 2.0;
	int its = 0;
	float scale = 2.0;
	float freq = 1.0;
	float ampl = 1.0;
	for (int i = 0; i < 32; i++) {
		freq *= scale;
		ampl *= persistence;
		total += interpolateNoise(freq*x, freq*y, freq*z)*ampl;
	}
	return total;
}


__global__ void triangleSimKernel(int numVerts, glm::mat4 *transforms, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	int triangleIdx = i / 3;
	int firstPos = triangleIdx * 3;
	glm::vec3 triangleCenter = glm::vec3(0.f);
	for (int j = 0; j < 3; j++) {
		auto &v = in[firstPos + j];
		glm::vec3 p = xm2vec3(v.position);
		triangleCenter += p;
	}
	triangleCenter /= 3.0f;
	//for (int j = 0; j < 3; ++j) {
	auto &v = in[i];
	glm::vec3 pos = xm2vec3(v.position);
	glm::vec3 nor = xm2vec3(v.normal);

	if (nor[1] == 1.0) {
		glm::vec3 p = twist(triangleCenter, time / 10.f);
		glm::mat4 transform = glm::translate(p);
		pos = glm::vec3(transform * glm::vec4(pos, 1.0f));
		out[i] = makeCUDAVertex(pos, nor);
	}
	else {
		glm::vec3 p = twist(pos, time / 10.f);
		glm::mat4 transform = glm::mat4(1.f);
		pos = glm::vec3(transform * glm::vec4(p, 1.0f));

		out[i] = makeCUDAVertex(pos, nor);
	}
	
	//}

}