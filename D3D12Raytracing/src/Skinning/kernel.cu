#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cudaSkinning.h"

#include <hip/hip_runtime.h>
#define GLM_FORCE_PURE
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <stdio.h>

#define PI 3.14159265358979323846

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time);
__global__ void skinKernel(int numVerts, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *in, cudaVertex *out, const float time);
void cudaFakeSkin(int numVerts, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **) &dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **) &dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	fakeSkinKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);
	hipFree(dv_idata);
	hipFree(dv_odata);
}

void cudaSkin(int numVerts, int numTransforms, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **)&dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **)&dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	glm::mat4 *dv_transforms;
	hipMalloc((void **)&dv_transforms, numTransforms * sizeof(glm::mat4));
	hipMemcpy(dv_transforms, transforms, numTransforms * sizeof(glm::mat4), hipMemcpyHostToDevice);

	cudaVertexBoneData *dv_bones;
	hipMalloc((void **)&dv_bones, numVerts * sizeof(cudaVertexBoneData));
	hipMemcpy(dv_bones, bones, numVerts * sizeof(cudaVertexBoneData), hipMemcpyHostToDevice);

	skinKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_transforms, dv_bones, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);

	hipFree(dv_idata);
	hipFree(dv_odata);
	hipFree(dv_transforms);
	hipFree(dv_bones);
}


__forceinline__
__device__ __host__
double convertToRadians(double deg) {
	return deg * (PI / 180.);
}

__forceinline__
__device__ __host__
glm::vec3 xm2vec3(XMFLOAT3 other) {
	return glm::vec3(other.x, other.y, other.z);
}

__forceinline__
__device__ __host__
glm::mat4 xm2mat4(XMFLOAT4X4 other) {
	float aaa[16];
	aaa[0] = other._11;
	aaa[1] = other._21;
	aaa[2] = other._31;
	aaa[3] = other._41;

	aaa[4] = other._12;
	aaa[5] = other._22;
	aaa[6] = other._32;
	aaa[7] = other._42;

	aaa[8] = other._13;
	aaa[9] = other._23;
	aaa[10] = other._33;
	aaa[11] = other._43;

	aaa[12] = other._14;
	aaa[13] = other._24;
	aaa[14] = other._34;
	aaa[15] = other._44;
	return glm::make_mat4(aaa);
}

__forceinline__
__device__ __host__
cudaVertex makeCUDAVertex(glm::vec3 position, glm::vec3 normal) {
	cudaVertex ret;
	ret.position.x = position.x;
	ret.position.y = position.y;
	ret.position.z = position.z;
	ret.normal.x = normal.x;
	ret.normal.y = normal.y;
	ret.normal.z = normal.z;
	return ret;
}

__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v = in[i];
	glm::vec3 pos = xm2vec3(v.position);
	glm::vec3 nor = xm2vec3(v.normal);
	auto rot = glm::rotate((time / 24.f) * 360.f, glm::vec3(0, 1, 0));
	pos = glm::vec3(rot * glm::vec4(pos, 1.f));
	out[i] = makeCUDAVertex(pos, nor);
}

__global__ void skinKernel(int numVerts, glm::mat4 *transforms, cudaVertexBoneData *bones, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v = in[i];
	auto &bone = bones[i];
	glm::vec3 pos = xm2vec3(v.position);
	glm::vec3 nor = xm2vec3(v.normal);
	glm::mat4 transform(0.f);
	float totalWeight = 0.f;
	for (int k = 0; k < 4; k++) {
		int ID = bone.IDs[k];
		float weight = bone.Weights[k];
		totalWeight += weight;
		glm::mat4 mat = (transforms[ID]);
		transform += mat * weight;
	}

	pos = glm::vec3(transform * glm::vec4(pos, 1.0f));
	nor = glm::vec3(transform * glm::vec4(nor, 0.0f));

	out[i] = makeCUDAVertex(pos, nor);
}