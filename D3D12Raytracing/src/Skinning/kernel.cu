#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cudaSkinning.h"

#include <hip/hip_runtime.h>
#define GLM_FORCE_PURE
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/transform.hpp>

#include <stdio.h>

#define PI 3.14159265358979323846


__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time);

void cudaFakeSkin(int numVerts, cudaVertex *vertsIn, cudaVertex *vertsOut, const float time) {
	int blockSize = 128;
	dim3 blocksPerGrid((numVerts + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	cudaVertex *dv_idata, *dv_odata;

	hipMalloc((void **) &dv_idata, numVerts * sizeof(cudaVertex));
	hipMalloc((void **) &dv_odata, numVerts * sizeof(cudaVertex));
	hipMemcpy(dv_idata, vertsIn, numVerts * sizeof(cudaVertex), hipMemcpyHostToDevice);

	fakeSkinKernel << <blocksPerGrid, threadsPerBlock >> > (numVerts, dv_idata, dv_odata, time);

	hipMemcpy(vertsOut, dv_odata, numVerts * sizeof(cudaVertex), hipMemcpyDeviceToHost);
	hipFree(dv_idata);
	hipFree(dv_odata);
}

__forceinline__
__device__ __host__
double convertToRadians(double deg) {
	return deg * (PI / 180.);
}

__forceinline__
__device__ __host__
glm::vec3 xm2vec3(XMFLOAT3 other) {
	return glm::vec3(other.x, other.y, other.z);
}

__forceinline__
__device__ __host__
cudaVertex makeCUDAVertex(glm::vec3 position, glm::vec3 normal) {
	cudaVertex ret;
	ret.position.x = position.x;
	ret.position.y = position.y;
	ret.position.z = position.z;
	ret.normal.x = normal.x;
	ret.normal.y = normal.y;
	ret.normal.z = normal.z;
	return ret;
}

__global__ void fakeSkinKernel(int numVerts, cudaVertex *in, cudaVertex *out, const float time) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i >= numVerts) { return; }
	auto &v = in[i];
	glm::vec3 pos = xm2vec3(v.position);
	glm::vec3 nor = xm2vec3(v.normal);
	auto rot = glm::rotate((time / 24.f) * 360.f, glm::vec3(0, 1, 0));
	pos = glm::vec3(rot * glm::vec4(pos, 1.f));
	out[i] = makeCUDAVertex(pos, nor);
}
